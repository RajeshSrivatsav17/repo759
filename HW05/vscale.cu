#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include "vscale.cuh"
__global__ void vscale(const float *a, float *b, unsigned int n)
{
    int index = (blockIdx.x*blockDim.x) +threadIdx.x;
    if(n < index)
        return;

    b[index] = a[index]*b[index];
}