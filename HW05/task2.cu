#include<iostream>
#include<hip/hip_runtime.h>
#include <random>

__global__ void kernel(int *data, int a)
{
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    data[index] = a * threadIdx.x + blockIdx.x;
}

int main()
{
    const int numElems = 16;
    int hA[numElems],*dA;
    int a = std::rand();
    hipMalloc((void**)&dA, sizeof(int) * numElems);
    hipMemset(dA, 0, numElems * sizeof(int));
    kernel<<<2,8>>>(dA,a);
    hipMemcpy(hA, dA, sizeof(int) * numElems, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i=0;i<numElems;i++){
        printf("%d ", hA[i]);
    }
    printf("\n");

    return 0;
}
