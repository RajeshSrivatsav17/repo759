#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include <cstdlib>
#include <random>
#include <ctime>
#include "vscale.cuh"

void genRandomFloat(float * arr, std::size_t size, float min, float max){
    std::srand(std::time(0));
    for(long unsigned int i =0;i<size;i++){
        arr[i] = min + (((float)rand())/(float)RAND_MAX)*(max-min);
    }
}

int main(int argc, char *argv[]){
    if(argc < 2)
    {
        std::cout<<"Provide a valid number N\n";
        return -1;
    }
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::size_t n = (std::size_t)std::stol(argv[1]);
    int numBlocks = 0;
    int numThreads = 512;
    float *d_a, *d_b;
    float *h_a = new float[n]; 
    float *h_b = new float[n];

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    genRandomFloat(h_a, n, -10, 10);
    genRandomFloat(h_b, n, 0, 1);

    numBlocks = (n+numThreads-1) / numThreads;
    // Copy data from host to device
    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);
    vscale<<<numBlocks,numThreads>>>(d_a,d_b,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(h_b, d_b, sizeof(int) * n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout<<ms << "\n" << h_b[0] << "\n" << h_b[n-1] << "\n";
    return 0;
}
