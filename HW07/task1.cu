#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>
#include<ctime>
#include "matmul.cuh"

float genRandomfloat(float min, float max){
    return (min + (((float)std::rand())/(float)RAND_MAX)*(max-min));
}

double genRandomdouble(double min, double max){
    return (min + (((double)std::rand())/(double)RAND_MAX)*(max-min));
}

int genRandomInt(int min, int max) {
    return min + std::rand() % (max - min + 1);
}

void assignRandomNumberToInputArraysFloat(float * arr, std::size_t n){
    float random_num = 0.0;
    for(long unsigned int i =0;i<n;i++){
        random_num = genRandomfloat(-1,1);
        arr[i] = random_num;
    }  
}

void assignRandomNumberToInputArraysDouble(double * arr, std::size_t n){
    double random_num = 0.0;
    for(long unsigned int i =0;i<n;i++){
        random_num = genRandomdouble(-1,1);
        arr[i] = random_num;
    }  
}

void assignRandomNumberToInputArraysInt(int * arr, std::size_t n){
    int random_num = 0.0;
    for(long unsigned int i =0;i<n;i++){
        random_num = genRandomInt(-1,1);
        arr[i] = random_num;
    }  
}

int main(int argc, char* argv[]) {
    if(argc < 3)
    {
        std::cout <<"Please provide n and t arguments\n";
        return 0;
    }
    std::srand(std::time(0));

    int n = std::atoi(argv[1]);
    int block_dim = std::atoi(argv[2]);
    int block_dim_per_row_or_col = std::sqrt(block_dim);
    int size = n*n;

    int* A1 = new int[size];
    int* B1 = new int[size];
    int* C1 = new int[size];

    float* A2 = new float[size];
    float* B2 = new float[size];
    float* C2 = new float[size];

    double* A3 = new double[size];
    double* B3 = new double[size];
    double* C3 = new double[size];

    int* Ad1;
    int* Bd1;
    int* Cd1;

    float* Ad2;
    float* Bd2;
    float* Cd2;

    double* Ad3;
    double* Bd3;
    double* Cd3;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&Ad1, size* sizeof(int));
    hipMalloc((void**)&Bd1, size* sizeof(int));
    hipMalloc((void**)&Cd1, size* sizeof(int));

    hipMalloc((void**)&Ad2, size* sizeof(float));
    hipMalloc((void**)&Bd2, size* sizeof(float));
    hipMalloc((void**)&Cd2, size* sizeof(float));

    hipMalloc((void**)&Ad3, size* sizeof(double));
    hipMalloc((void**)&Bd3, size* sizeof(double));
    hipMalloc((void**)&Cd3, size* sizeof(double));

    assignRandomNumberToInputArraysInt(A1, size);
    assignRandomNumberToInputArraysInt(B1, size);
    assignRandomNumberToInputArraysFloat(A2, size);
    assignRandomNumberToInputArraysFloat(B2, size);
    assignRandomNumberToInputArraysDouble(A3, size);
    assignRandomNumberToInputArraysDouble(B3, size);

    hipMemcpy(Ad1, A1, size* sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(Bd1, B1, size* sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(Ad2, A2, size* sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(Bd2, B2, size* sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(Ad3, A3, size* sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(Bd3, B3, size* sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(start);
    matmul_1(Ad1, Bd1, Cd1, n, block_dim_per_row_or_col);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(C1, Cd1, size* sizeof(int), hipMemcpyDeviceToHost);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout<< C1[0] << "\n" << C1[size-1] <<"\n"<<ms << "\n";

    hipEventRecord(start, 0);
    hipEventRecord(stop, 0);

    hipEventRecord(start);
    matmul_2(Ad2, Bd2, Cd2, n, block_dim_per_row_or_col);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(C2, Cd2, size* sizeof(float), hipMemcpyDeviceToHost);

    hipEventElapsedTime(&ms, start, stop);

    std::cout<< C2[0] << "\n" << C2[size-1] <<"\n"<<ms << "\n";

    hipEventRecord(start, 0);
    hipEventRecord(stop, 0);

    hipEventRecord(start);
    matmul_3(Ad3, Bd3, Cd3, n, block_dim_per_row_or_col);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(C3, Cd3, size* sizeof(double), hipMemcpyDeviceToHost);

    std::cout<< C3[0] << "\n" << C3[size-1] <<"\n"<<ms << "\n";

    hipEventRecord(start, 0);
    hipEventRecord(stop, 0);
    delete[] A1; delete[] B1; delete[] C1;
    delete[] A2; delete[] B2; delete[] C2;
    delete[] A3; delete[] B3; delete[] C3;
    
    hipFree(Ad1); hipFree(Bd1); hipFree(Cd1);
    hipFree(Ad2); hipFree(Bd2); hipFree(Cd2);
    hipFree(Ad3); hipFree(Bd3); hipFree(Cd3);

    return 0;
}