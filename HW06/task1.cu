#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

float genRandomfloat(float min, float max){
    return (min + (((float)std::rand())/(float)RAND_MAX)*(max-min));
}

void assignRandomNumberToInputArrays(float * arr, std::size_t n){
    float random_num = 0.0;
    for(long unsigned int i =0;i<n;i++){
        random_num = genRandomfloat(-1,1);
        arr[i] = random_num;
    }  
}

int main(int argc, char* argv[]) {
    if(argc < 3)
    {
        std::cout <<"Please provide n and t arguments\n";
        return 0;
    }
    std::srand(std::time(0));

    int n = std::atoi(argv[1]);
    int threads_per_block = std::atoi(argv[2]);
    int size = n*n;

    float* A = new float[size];
    float* B = new float[size];
    float* C = new float[size];

    float* Ad;
    float* Bd;
    float* Cd;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&Ad, size* sizeof(float));
    hipMalloc((void**)&Bd, size* sizeof(float));
    hipMalloc((void**)&Cd, size* sizeof(float));

    assignRandomNumberToInputArrays(A, size);
    assignRandomNumberToInputArrays(B, size);

    hipMemcpy(Ad, A, size* sizeof(float), hipMemcpyHostToDevice); 
    hipMemcpy(Bd, B, size* sizeof(float), hipMemcpyHostToDevice);

    int block_dim = sqrt(threads_per_block);

    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (n + dimBlock.y - 1) / dimBlock.y);

    hipEventRecord(start);
    matmul_kernel<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipDeviceSynchronize();
    hipMemcpy(C, Cd, size* sizeof(float), hipMemcpyDeviceToHost);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    std::cout<< C[size-1] <<"\n"<<ms << "\n";

    return 0;
}
